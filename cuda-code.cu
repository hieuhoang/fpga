#include <cstddef>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cuda-code.h"
#include "cuda-matrix.h"
#include "types-cuda.h"

using namespace std;

void runCuda(HostMatrix<MaxY_type> &maxY, const HostMatrix<float> &W, const HostMatrix<float> &X, const HostMatrix<float> &B)
{ 
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "hipblasCreate initialization failed" << std::endl;
    abort();
  }

  hipblasOperation_t opA = HIPBLAS_OP_N;
  hipblasOperation_t opB = HIPBLAS_OP_N;

  int m = 85000;
  int n = 640;
  int k = 512;

  int lda = 85000;
  int ldb = 512;
  int ldc = 85000;

  const float alpha = 1;
  const float beta = 0;

  CudaMatrix<float> cudaX(X);

  const float *Aptr;
  const float *Bptr;
  float *Cptr;

  HANDLE_ERROR_CUBLAS(hipblasSgemm(handle, opA, opB,
                      m, n, k,
                      &alpha,
                      Aptr, lda,
                      Bptr, ldb,
                      &beta,
                      Cptr, ldc));

}


