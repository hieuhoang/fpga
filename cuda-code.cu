#include <cstddef>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cuda-code.h"
#include "cuda-matrix.h"
#include "types-cuda.h"

using namespace std;

void runCuda(HostMatrix<MaxY_type> &maxY, const HostMatrix<float> &W, const HostMatrix<float> &X, const HostMatrix<float> &B)
{ 
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "hipblasCreate initialization failed" << std::endl;
    abort();
  }

  hipblasOperation_t opA = HIPBLAS_OP_N;
  hipblasOperation_t opB = HIPBLAS_OP_N;

  int m = 85000;
  int n = 640;
  int k = 512;

  int lda = 85000;
  int ldb = 512;
  int ldc = 85000;

  const float alpha = 1;
  const float beta = 0;

  CudaMatrix<float> cudaW(W);
  CudaMatrix<float> cudaX(X);
  CudaMatrix<float> cudaB(B);
  CudaMatrix<float> cudaY(85000, 640);

  HANDLE_ERROR_CUBLAS(hipblasSgemm(handle, opA, opB,
                      m, n, k,
                      &alpha,
                      cudaW.data(), lda,
                      cudaX.data(), ldb,
                      &beta,
                      cudaY.data(), ldc));

}


