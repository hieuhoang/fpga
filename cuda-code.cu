#include <cstddef>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cuda-code.h"

using namespace std;

void HandleError(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    std::cerr << "ERROR: " << hipGetErrorString(err) << " in " << file << " at line " << line << std::endl;
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void HandleErrorCublas(hipblasStatus_t err, const char *file, int line ) {
  if (err != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "CUBLAS ERROR: " << err << " in " << file << " at line " << line << std::endl;
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR_CUBLAS( err ) (HandleErrorCublas( err, __FILE__, __LINE__ ))

void runCuda()
{ 
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "hipblasCreate initialization failed" << std::endl;
    abort();
  }

  hipblasOperation_t opA = HIPBLAS_OP_N;
  hipblasOperation_t opB = HIPBLAS_OP_N;

  int m = 85000;
  int n = 640;
  int k = 512;

  int lda = 85000;
  int ldb = 512;
  int ldc = 85000;

  const float alpha = 1;
  const float beta = 0;
  const float *A;
  const float *B;
  float *C;

  HANDLE_ERROR( hipMalloc(&A, 85000 * 512 * sizeof(float)) );
  HANDLE_ERROR( hipMalloc(&B, 512 * 640 * sizeof(float)) );
  HANDLE_ERROR( hipMalloc(&C, 85000 * 640 * sizeof(float)) );

  HANDLE_ERROR_CUBLAS(hipblasSgemm(handle, opA, opB,
                      m, n, k,
                      &alpha,
                      A, lda,
                      B, ldb,
                      &beta,
                      C, ldc));

}


