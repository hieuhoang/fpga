#include <cstddef>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cuda-code.h"

using namespace std;

void HandleErrorCublas(hipblasStatus_t err, const char *file, int line ) {
  if (err != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "CUBLAS ERROR: " << err << " in " << file << " at line " << line << std::endl;
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR_CUBLAS( err ) (HandleErrorCublas( err, __FILE__, __LINE__ ))

void runCuda()
{ 
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "hipblasCreate initialization failed" << std::endl;
    abort();
  }

  hipblasOperation_t opA = HIPBLAS_OP_N;
  hipblasOperation_t opB = HIPBLAS_OP_N;

  int m = 7;
  int n = 7;
  int k = 7;

  int lda = 4;
  int ldb = 6;
  int ldc = 7;

  const float *alpha = NULL;
  const float *beta = NULL;
  const float *A = NULL;
  const float *B = NULL;
  float *C = NULL;

  HANDLE_ERROR_CUBLAS(hipblasSgemm(handle, opB, opA,
                      n, m, k,
                      alpha,
                      B, ldb,
                      A, lda,
                      beta,
                      C, ldc));

}


