#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hipblas.h>
#include <mma.h>
#include "tensor-cores.h"
#include "types-cuda.h"

using namespace nvcuda;


// Must be multiples of 16 for wmma code to work
//#define MATRIX_M 16384
//#define MATRIX_N 16384
//#define MATRIX_K 16384
#define MATRIX_M 160
#define MATRIX_N 160
#define MATRIX_K 160


// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

////////////////////////////////////////////////////////////////////////////////////////////////


// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.
__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K, float alpha, float beta) {
   // Leading dimensions. Packed with no transpositions.
   int lda = M;
   int ldb = K;
   int ldc = M;

   // Tile using a 2D grid
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

   wmma::fill_fragment(acc_frag, 0.0f);

   // Loop over k
   for (int i = 0; i < K; i += WMMA_K) {
      int aRow = warpM * WMMA_M;
      int aCol = i;

      int bRow = i;
      int bCol = warpN * WMMA_N;

      // Bounds checking
      if (aRow < M && aCol < K && bRow < K && bCol < N) {
         // Load the inputs
         wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
         wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

         // Perform the matrix multiplication
         wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

      }
   }

   // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
   int cRow = warpM * WMMA_M;
   int cCol = warpN * WMMA_N;

   if (cRow < M && cCol < N) {
      wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);


      for(int i=0; i < c_frag.num_elements; i++) {
         c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
      }

      // Store the output
      wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
   }
}


__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

////////////////////////////////////////////////////////////////////////////////////////////////

void RunTensorCores()
{
  float *a_fp32;
   float *b_fp32;
   half *a_fp16;
   half *b_fp16;

   float *c;
   float *c_cublas;
   float *c_wmma;

   float *c_host_cublas;
   float *c_host_wmma;

   hiprandGenerator_t gen;
   hipblasHandle_t cublasHandle;

   hipEvent_t startWMMA;
   hipEvent_t stopWMMA;

   hipEvent_t startcublas;
   hipEvent_t stopcublas;

   HANDLE_ERROR(hipEventCreate(&startWMMA));
   HANDLE_ERROR(hipEventCreate(&stopWMMA));

   HANDLE_ERROR(hipEventCreate(&startcublas));
   HANDLE_ERROR(hipEventCreate(&stopcublas));


   HANDLE_ERROR_CUBLAS(hipblasCreate(&cublasHandle));

   // Use tensor cores
   HANDLE_ERROR_CUBLAS(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

   HANDLE_ERROR(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
   HANDLE_ERROR(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
   HANDLE_ERROR(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
   HANDLE_ERROR(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

   HANDLE_ERROR(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(float)));
   HANDLE_ERROR(hipMalloc((void**)&c_cublas, MATRIX_M * MATRIX_N * sizeof(float)));
   HANDLE_ERROR(hipMalloc((void**)&c_wmma, MATRIX_M * MATRIX_N * sizeof(float)));

   c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
   c_host_wmma = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

   HANDLE_ERROR_CURAND(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
   HANDLE_ERROR_CURAND(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

   HANDLE_ERROR_CURAND(hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
   HANDLE_ERROR_CURAND(hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));

   // hiprand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K);
   convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N);

   HANDLE_ERROR_CURAND(hiprandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));

   HANDLE_ERROR_CURAND(hiprandDestroyGenerator(gen));

   HANDLE_ERROR(hipMemcpy(c_cublas, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));
   HANDLE_ERROR(hipMemcpy(c_wmma, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));

   float alpha = 2.0f;
   float beta = 2.0f;


   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

   // First: using WMMA
   dim3 gridDim;
   dim3 blockDim;

   // blockDim.x must be a multple of warpSize
   // 128x4 means we have 16 warps and a block computes a 64x64 output tile
   blockDim.x = 128;
   blockDim.y = 4;

   gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
   gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

   printf("Running with wmma...\n");
   HANDLE_ERROR(hipEventRecord(startWMMA));
   wmma_example <<< gridDim, blockDim >>> (a_fp16, b_fp16, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   HANDLE_ERROR(hipEventRecord(stopWMMA));



   // Now using cuBLAS
   printf("Running with cuBLAS...\n");
   HANDLE_ERROR(hipEventRecord(startcublas));
   HANDLE_ERROR_CUBLAS(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_M, MATRIX_N, MATRIX_K,
                &alpha,
                a_fp16, HIP_R_16F, MATRIX_M,
                b_fp16, HIP_R_16F, MATRIX_K,
                &beta,
                c_cublas, HIP_R_32F, MATRIX_M,
                HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
   HANDLE_ERROR(hipEventRecord(stopcublas));

   // Error checking
   printf("\nChecking results...\n");
   HANDLE_ERROR(hipMemcpy(c_host_wmma, c_wmma, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   HANDLE_ERROR(hipMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));

   // 0.01% relative tolerance. 1e-5 absolute tolerance.
   int errors = 0;
   for (int i = 0; i < MATRIX_M * MATRIX_N; i++) {
      float v1 = c_host_wmma[i];
      float v2 = c_host_cublas[i];
      if (v1 / v2 > 1.0001 || v2 / v1 > 1.0001 || abs(v1 - v2) > 1e-5) {
         errors++;
         if (errors < 10) printf("%f %f\n", v1, v2);
      }
   }

   if (errors > 0) {
      printf("WMMA does not agree with cuBLAS! %d errors!\n", errors);
   }
   else {
      printf("Results verified: cublas and WMMA agree.\n\n");
      float wmmaTime;
      float cublasTime;
      HANDLE_ERROR(hipEventSynchronize(stopWMMA));
      HANDLE_ERROR(hipEventSynchronize(stopcublas));
      HANDLE_ERROR(hipEventElapsedTime(&wmmaTime, startWMMA, stopWMMA));
      HANDLE_ERROR(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
      printf("wmma took %fms\n", wmmaTime);
      printf("cublas took %fms\n", cublasTime);

      printf("\nFor a faster code using wmma you should check out the cudaTensorCoreGemm sample in the CUDA Toolkit.\nThis code was written as a demo only!\n\n");
   }


   HANDLE_ERROR(hipEventDestroy(startWMMA));
   HANDLE_ERROR(hipEventDestroy(stopWMMA));

   HANDLE_ERROR(hipEventDestroy(startcublas));
   HANDLE_ERROR(hipEventDestroy(stopcublas));

   HANDLE_ERROR(hipFree(a_fp32));
   HANDLE_ERROR(hipFree(b_fp32));
   HANDLE_ERROR(hipFree(a_fp16));
   HANDLE_ERROR(hipFree(b_fp16));

   HANDLE_ERROR(hipFree(c));
   HANDLE_ERROR(hipFree(c_cublas));
   HANDLE_ERROR(hipFree(c_wmma));

   free(c_host_cublas);
   free(c_host_wmma);

   HANDLE_ERROR(hipDeviceReset());

}
