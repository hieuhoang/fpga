#include "hip/hip_runtime.h"
#include <cstddef>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cuda-code.h"
#include "cuda-matrix.h"
#include "cuda-matrix-wrapper.h"
#include "types-cuda.h"

using namespace std;

__global__
void gCalcMax(CudaMatrixWrapper<MaxY_type> out, const CudaMatrixWrapper<float> in)
{
  assert(out.dim(1) == in.dim(1));
  for (unsigned col = 0; col < in.dim(1); ++col) {
    unsigned maxIndex = 0;
    float value = in(0, col);

    for (unsigned row = 1; row < in.dim(0); ++row) {
      float val = in(row, col);
      if (val > value) {
        value = val;
        maxIndex = row;
      }
    }

    MaxY_type &ele = out[col];
    ele.value = value;
    ele.index = maxIndex;
  }
}

void RunCuda(HostMatrix<MaxY_type> &maxY, const HostMatrix<float> &W, const HostMatrix<float> &X, const HostMatrix<float> &B)
{ 
  hipblasHandle_t handle;
  hipblasStatus_t stat;
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "hipblasCreate initialization failed" << std::endl;
    abort();
  }

  hipblasOperation_t opA = HIPBLAS_OP_N;
  hipblasOperation_t opB = HIPBLAS_OP_N;

  int m = 85000;
  int n = 640;
  int k = 512;

  int lda = 85000;
  int ldb = 512;
  int ldc = 85000;

  const float alpha = 1;
  const float beta = 0;

  CudaMatrix<float> cudaW(W);
  CudaMatrix<float> cudaX(X);
  CudaMatrix<float> cudaB(B);
  CudaMatrix<float> cudaY(85000, 640);

  HANDLE_ERROR_CUBLAS(hipblasSgemm(handle, opA, opB,
                      m, n, k,
                      &alpha,
                      cudaW.data(), lda,
                      cudaX.data(), ldb,
                      &beta,
                      cudaY.data(), ldc));

  CudaMatrix<MaxY_type> cudaMaxY(1, 640);
  gCalcMax<<<1,1>>>(cudaMaxY, cudaY);

  hipDeviceSynchronize();

  cudaMaxY.CopyTo(maxY);
}


